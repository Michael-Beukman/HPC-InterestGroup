#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include "./common/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#define IDX(y, x) ((y) * W + (x))
#define BLOCK_SIZE 16

__device__ int neighbours_shared(int x, int y, int W, int H, bool* cells) {
    int N = 0;
    int tx, ty;
    // for all neighbours
    #pragma unroll
    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            // not the cell itself
            if (!i && !j) continue;
            tx = x + i;
            ty = y + j;

            // add if the cell is active
            int idx = ty * W + tx;
            N += cells[idx];
        }
    }
    return N;
}


void print(int W, int H, bool* cells) {
    for (int y = 0; y < H; ++y) {
        for (int x = 0; x < W; ++x) {
            if (cells[y * W + x])
                printf("#");
            else 
                printf(".");
        }
        printf("\n");
    }
}

// game of life kernel in CUDA:

__global__ void game_of_life(bool* d_Cells, bool* d_Buffer, int w, int h, int A, int B, int C, int internal_iters) {
    __shared__ bool data_of_the_things[(BLOCK_SIZE + 2) * (BLOCK_SIZE + 2)]; // copy data now

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int testx = min(BLOCK_SIZE, w);
    int testy = min(BLOCK_SIZE, h);

    int TEST = BLOCK_SIZE + 2;

    if (x < w && y < h){
        int my_tx = threadIdx.x + 1;
        int my_ty = threadIdx.y + 1;
        int idx = y * w + x;

        // copy main data

        data_of_the_things[my_ty * TEST + my_tx] = d_Cells[idx];


        if (threadIdx.x == 0){
            data_of_the_things[my_ty * TEST + 0] = d_Cells[y * w + (x - 1 + w) % w];
        }
        if (threadIdx.y == 0){
            data_of_the_things[0 * TEST + my_tx] = d_Cells[((y - 1 + h) % h) * w + x];
        }
        if (threadIdx.x == testx - 1){
            data_of_the_things[my_ty * TEST + testx + 1] = d_Cells[y * w + (x + 1) % w];
        }
        if (threadIdx.y == testy - 1){
            data_of_the_things[(testy+1) * TEST + my_tx] = d_Cells[((y + 1) % h) * w + x];
        }

        // now do 4 corners
        if (threadIdx.x == 0 && threadIdx.y == 0){
            data_of_the_things[0 * TEST + 0] = d_Cells[((y - 1 + h) % h) * w + (x - 1 + w) % w];
        }
        if (threadIdx.x == testx - 1 && threadIdx.y == 0){
            data_of_the_things[0 * TEST + testx + 1] = d_Cells[((y - 1 + h) % h) * w + (x + 1) % w];
        }
        if (threadIdx.x == 0 && threadIdx.y == testy - 1){
            data_of_the_things[(testy+1) * TEST + 0] = d_Cells[((y + 1) % h) * w + (x - 1 + w) % w];
        }
        if (threadIdx.x == testx - 1 && threadIdx.y == testy - 1){
            data_of_the_things[(testy+1) * TEST + testx + 1] = d_Cells[((y + 1) % h) * w + (x + 1) % w];
        }


        __syncthreads(); // synchronise here.

        int N = neighbours_shared(my_tx, my_ty, TEST, TEST, data_of_the_things);
        bool is_on = d_Cells[idx];
        bool new_val = (is_on && (N >= A && N <= B)) || (!is_on && N == C);

        d_Buffer[idx] = new_val;
    }
}

int main(int argc, char** argv) {
    setvbuf(stdout, NULL, _IOFBF, 16384 * 16); // https://stackoverflow.com/a/65020999

    int w, h, n, m, A, B, C;
    scanf("%d %d %d %d %d %d %d\n", &w, &h, &n, &m, &A, &B, &C);
    bool* cells  = (bool*) malloc(w * h * sizeof(bool));
    bool* buffer = (bool*) malloc(w * h * sizeof(bool));

    bool* d_Cells;
    bool* d_Buffer;
    
    checkCudaErrors(hipMalloc(&d_Cells, w * h * sizeof(bool)));
    checkCudaErrors(hipMalloc(&d_Buffer, w * h * sizeof(bool)));
    

    char c;
    // now read the grid
    for (int i = 0; i < h; ++i) {
        for (int j = 0; j < w; ++j) {
            scanf("%c", &c);
            if (c == '#') 
                cells[i * w + j] = 1;
            else
                cells[i * w + j] = 0;
        }
        scanf("%c", &c); // newlines
    }


    // copy data to d_Cells and d_Buffer;
    checkCudaErrors(hipMemcpy(d_Cells, cells, w * h * sizeof(bool), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_Buffer, d_Cells, w * h * sizeof(bool), hipMemcpyDeviceToDevice));

    int i = 0;
    float angle = 0;
    // Used to get a consistent frame rate
    for (int step=0; step < n; ++step){
        int step_one_indexed = step + 1;
        // update the new array
        dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

        dim3 num_blocks((w + block_size.x - 1) / block_size.x, (h + block_size.y - 1) / block_size.y);
        game_of_life<<<num_blocks, block_size>>>(d_Cells, d_Buffer, w, h, A, B, C, m);
        // Swap the vectors so that the updated one is drawn in the next frame.
        bool* temp = d_Buffer;
        d_Buffer = d_Cells;
        d_Cells = temp;

        if (step == 0 || (step + 1) % m == 0 || step == n - 1) {
            // copy d_Cells to cells:
            checkCudaErrors(hipMemcpy(cells, d_Cells, w * h * sizeof(bool), hipMemcpyDeviceToHost));
            print(w, h, cells);
        }
    }
    free(cells);
    free(buffer);

    hipFree(d_Buffer);
    hipFree(d_Cells);
}